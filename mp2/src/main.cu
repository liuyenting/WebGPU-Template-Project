#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(
  float *A, float *B, float *C,
  int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns
) {
  //@@ Insert code to implement matrix multiplication here
  int column = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  // A_ik * B_kj = C_ij
  if ((column < numCColumns) && (row < numCRows)) {
    float sum = 0;
    for (int k = 0; k < numAColumns; k++) {
      sum += A[row * numAColumns + k] * B[k * numBColumns + column];
    }
    C[row * numCColumns + column] = sum;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  size_t sizeC = numCRows*numCColumns*sizeof(float);
  hostC = (float *)malloc(sizeC);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  size_t sizeA = numARows*numAColumns*sizeof(float);
  size_t sizeB = numBRows*numBColumns*sizeof(float);
  wbCheck(hipMalloc(&deviceA, sizeA));
  wbCheck(hipMalloc(&deviceB, sizeB));
  wbCheck(hipMalloc(&deviceC, sizeC));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  wbCheck(hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(32, 32);
  dim3 gridDim(
    ceil((float)numCColumns/32),
    ceil((float)numCRows/32)
  );
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<gridDim, blockDim>>>(
    deviceA, deviceB, deviceC,
    numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns
  );
  wbCheck(hipDeviceSynchronize());
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  wbCheck(hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  wbCheck(hipFree(deviceA));
  wbCheck(hipFree(deviceB));
  wbCheck(hipFree(deviceC));
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
